#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>
#include<string.h>
#include<vector>
#include<time.h>

int SIZE;
struct tree;

void print(int*);
int count_child(tree*, int);

#define maxthreads 1024

#include"AI.cu"

using namespace std;

void print(int* board){
	cout << "   |";
	for(int a = 0; a < SIZE; a++){
		cout << " " << a << " |";
	}
	cout << endl;
	for(int a = 0; a < SIZE+1; a++){
		cout << "----";
	}
	cout << endl;
	for(int a = 0; a < SIZE; a++){
		cout << " " << a << " |";
		for(int b = 0; b < SIZE; b++){
			switch((*(board+a) >> (2*b)) & 0x00000003){
				case 2:
					cout << " ● |";
					break;
				case 3:
					cout << " ○ |";
					break;
				default:
					cout << " . |";
					break;
			}
		}
		cout << endl;
		for(int b = 0; b < SIZE+1; b++){
			cout << "----";
		}
		cout << endl;
	}
}

int* Init(){
	int* board = new int[SIZE];
	*(board+SIZE/2-1) += 3*pow(4, SIZE/2-1) + 2*pow(4, SIZE/2);
	*(board+SIZE/2) += 2*pow(4, SIZE/2-1) + 3*pow(4, SIZE/2);
	print(board);
	return board;
}

int check(int* board, int posi_x, int posi_y, int color, bool fswitch){
	int return_val = 0;
	if((board[posi_x] >> (2*posi_y) & 0x00000003) != 0){
		return 0;
	}
	for(int dire = 0; dire < 9; dire++){
		if(dire == 4){
			continue;
		}
		int sigmax = (dire/3)-1;
		int sigmay = (dire%3)-1;
		int x_ = posi_x + sigmax;
		int y_ = posi_y + sigmay;
		while(x_ != -1 && x_ != SIZE && y_ != -1 && y_ != SIZE){
			int target = (board[x_] >> (2*y_) & 0x00000003);
			if(target == 3-color){
				x_ += sigmax;
				y_ += sigmay;
				continue;
			}
			else if(target == 0){
				break;
			}
			else if(target == 2+color){
				int x__ = posi_x + sigmax;
				int y__ = posi_y + sigmay;
				while(x__ != x_ || y__ != y_){
					if(fswitch){
						board[x__] += (2*color-1)*pow(4, y__);
					}
					x__ += sigmax;
					y__ += sigmay;
					return_val++;
				}
				break;
			}
			else{
				cout << "Error!" << x_ << y_ << endl;
				exit(EXIT_FAILURE);
			}
		}
	}
	if(fswitch){
		board[posi_x] += (color+2)*pow(4, posi_y);
	}
	return return_val;
}

int count_child(tree* root, int return_val){
	if(root->leaf != NULL){
		return ++return_val;
	}
	for(int a = 0; a < 4; a++){
		if(root->next_block[a] != NULL){
			return_val = count_child(root->next_block[a], return_val);
		}
	}
	return return_val;
}

int main(int argc, char* argv[]){
	srand(time(NULL));
	if(argc != 2){
		cout << "Wrong input! Invalid chessboard size." << endl;
		return 0;
	}
	else{
		int buf = 0;
		for(int a = 0; a < strlen(argv[1]); a++){
			if(argv[1][a] - '0' > 9 || argv[1][a] - '0' < 0){
				cout << "Wrong input! Invalid chessboard size." << endl;
				return 0;
			}
			else{
				buf *= 10;
				buf += argv[1][a] - '0';
			}
		}
		
		if(buf%2){
			cout << "Wrong input! Even numbers only!" << endl;
			return 0;
		}
		
		SIZE = buf;
	}
	char choise;
	cin >> choise;
	int* board = Init();
	tree* root = new tree;
	int* search_data;
	hipMallocManaged(&search_data, sizeof(int)*SIZE);
	memcpy(search_data, board, sizeof(int)*SIZE);
	make_leaf(root, board);
	search_recu(1, search_data, 0, root, 0);
	/*
	   cout << endcount << endl;
	   cout << error_count << endl;
	   cout << count_child(root, 0) << endl;
	 */
	int chess_num[2] = {2, 2};
	int color = 0;
	int x, y;
	cout << chess_num[0] << " : " << chess_num[1] << endl;
	cout << "● 's turn: ";
	if(choise == '1'){
		while(1){
			vector<int> avamov;
			for(int posi = 0; posi < SIZE*SIZE; posi++){
				if(check(board, posi/SIZE, posi%SIZE, color, 0)){
					avamov.push_back(posi);
				}
			}
			while(avamov.size() != 0){
				cout << "(" << avamov.size() << " possible moves)"<< endl;
				for(int a = 0; a < avamov.size(); a++){
					cout << avamov[a] << " ";
				}
				cout << endl;
				cin >> x >> y;
				while(x >= SIZE || x <= -1 || y >= SIZE || y <= -1){
					cout << "Wrong input! Out of range." << endl;
					cin >> x >> y;
				}
				int index = x*SIZE+y;
				bool lock = 1;
				for(int a = 0; a < avamov.size(); a++){
					if(index == avamov[a]){
						int sigma = check(board, x, y, color, 1);
						chess_num[color] += sigma + 1;
						chess_num[1-color] -= sigma;
						color = 1-color;
						lock = 0;
						break;
					}
				}
				if(lock){
					cout << "Wrong input! You can't flip any enemy chess." << endl;
					continue;
				}
				else{
					print(board);
					cout << chess_num[0] << " : " << chess_num[1] << endl;
					break;
				}
			}
			if(chess_num[0] + chess_num[1] == SIZE*SIZE){
				break;
			}
			if(!(avamov.size())){
				cout << "Can't make any move!\n\n" << endl;
				color = 1-color;
			}
			switch(color){
				case 0:
					cout << "● 's turn: ";
					break;
				case 1:
					cout << "○ 's turn: ";
					break;
			}
			continue;
		}
	}
	else if(choise == '2'){
		while(1){
			vector<int> avamov;
			for(int posi = 0; posi < SIZE*SIZE; posi++){
				if(check(board, posi/SIZE, posi%SIZE, color, 0)){
					avamov.push_back(posi);
				}
			}
			if(color == 0){
				while(avamov.size() != 0){
					cout << "(" << avamov.size() << " possible moves)"<< endl;
					for(int a = 0; a < avamov.size(); a++){
						cout << avamov[a] << " ";
					}
					cout << endl;
					cin >> x >> y;
					while(x >= SIZE || x <= -1 || y >= SIZE || y <= -1){
						cout << "Wrong input! Out of range." << endl;
						cin >> x >> y;
					}
					int index = x*SIZE+y;
					bool lock = 1;
					for(int a = 0; a < avamov.size(); a++){
						if(index == avamov[a]){
							int sigma = check(board, x, y, color, 1);
							chess_num[color] += sigma + 1;
							chess_num[1-color] -= sigma;
							color = 1-color;
							lock = 0;
							break;
						}
					}
					if(lock){
						cout << "Wrong input! You can't flip any enemy chess." << endl;
						continue;
					}
					else{
						print(board);
						cout << chess_num[0] << " : " << chess_num[1] << endl;
						break;
					}
				}
				if(chess_num[0] + chess_num[1] == SIZE*SIZE){
					break;
				}
				if(avamov.size() == 0){
					cout << "Can't make any move!\n\n" << endl;
					color = 1-color;
				}
				cout << "○'s turn: ";
				continue;
			}
			else{		//	AI move.
				if(avamov.size() != 0){
					int sigma;
					bool win = 0;
					for(int a = 0; a < avamov.size(); a++){
						int* board_cpy = new int[SIZE];
						memcpy(board_cpy, board, sizeof(int)*SIZE);
						sigma = check(board, avamov[a]/SIZE, avamov[a]%SIZE, 1, 1);
						if(get_leaf_pointer(root, board)->white_value == 1){
							win = 1;
							delete [] board_cpy;
							break;
						}
						memcpy(board, board_cpy, sizeof(int)*SIZE);
					}
					if(win){
						cout << "Meow." << endl;
					}
					else{
						sigma = check(board, avamov[rand()%avamov.size()]/SIZE, avamov[rand()%avamov.size()]%SIZE, 1, 1);
						cout << "Meow!" << endl;
					}
					chess_num[1] += sigma + 1;
					chess_num[0] -= sigma;
					print(board);
					color = 1-color;
				}
			}
			if(chess_num[0] + chess_num[1] == SIZE*SIZE){
				break;
			}
			if(!(avamov.size())){
				cout << "Can't make any move!\n\n" << endl;
				color = 1-color;
			}
			cout << "●'s turn :";
			continue;
		}
	}
	cout << "Game Over!\nFinal score:\t" << chess_num[0] << " : " << chess_num[1] << endl;
	if(chess_num[0] > chess_num[1]){
		cout << "Black wins!" << endl;
	}
	else if(chess_num[1] > chess_num[0]){
		cout << "White wins!" << endl;
	}
	else{
		cout << "It's a tie!" << endl;
	}
	return 0;
}
