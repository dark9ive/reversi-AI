#include "hip/hip_runtime.h"
//
//	AI.cu
//

using namespace std;
struct leaves;

struct tree{
	tree* next_block[4] = {NULL, NULL, NULL, NULL};
	leaves* leaf = NULL;
};
struct leaves{
	int black_value = -1;
	int white_value = -1;
};
int error_count = 0;
int endcount = 0;

int make_leaf(tree* root, int* board){
	for(int x = 0; x < SIZE; x++){
		for(int y = 0; y < SIZE; y++){
			if(root->next_block[board[x] >> (2*y) & 0x00000003] == NULL){
				root->next_block[board[x] >> (2*y) & 0x00000003] = new tree;
			}
			root = root->next_block[board[x] >> (2*y) & 0x00000003];
		}
	}
	if(root->leaf == NULL){
		root->leaf = new leaves;
		return 1;
	}
	else{
		return 0;
	}
}

__global__ void cuda_check(int* board, int* cuda_data_out, int SIZE, int color){
	board += SIZE*threadIdx.x;
	int board_cpy[8];
	memcpy(board_cpy, board, sizeof(int)*SIZE);
	int anscount = 0;
	int used;
	for(int counter = 0; counter < 2; counter++){
		used = 0;
		for(int index = 0; index < SIZE*SIZE; index++){
			int posi_x = index/SIZE;
			int posi_y = index%SIZE;
			int return_val = 0;
			if((board[posi_x] >> (2*posi_y) & 0x00000003) != 0){
				used++;
				continue;
			}
			for(int dire = 0; dire < 9; dire++){
				if(dire == 4){
					continue;
				}
				int sigmax = (dire/3)-1;
				int sigmay = (dire%3)-1;
				int x_ = posi_x + sigmax;
				int y_ = posi_y + sigmay;
				while(x_ != -1 && x_ != SIZE && y_ != -1 && y_ != SIZE){
					int target = (board[x_] >> (2*y_) & 0x00000003);
					if(target == 3-color){
						x_ += sigmax;
						y_ += sigmay;
						continue;
					}
					else if(target == 0){
						break;
					}
					else if(target == 2+color){
						int x__ = posi_x + sigmax;
						int y__ = posi_y + sigmay;
						while(x__ != x_ || y__ != y_){
							board[x__] += (2*color-1)*pow(4, y__);
							x__ += sigmax;
							y__ += sigmay;
							return_val++;
						}
						break;
					}
					else{
						printf("ERROR! %d %d %d\n", x_, y_, threadIdx.x);
						return;
					}
				}
			}
			if(return_val){
				board[posi_x] += (color+2)*pow(4, posi_y);
				memcpy(cuda_data_out+(SIZE*SIZE*SIZE+1)*threadIdx.x+anscount*SIZE+1, board, sizeof(int)*SIZE);
				memcpy(board, board_cpy, sizeof(int)*SIZE);
				anscount++;
			}
		}
		if(anscount == 0){
			if(used == SIZE*SIZE || counter == 1){
				break;
			}
			color = 1 - color;
			continue;
		}
		if(counter == 1){
			anscount = 0;
			memcpy(cuda_data_out+(SIZE*SIZE*SIZE+1)*threadIdx.x+anscount*SIZE+1, board, sizeof(int)*SIZE);
			anscount = 1;
		}
		break;
	}
	*(cuda_data_out+(SIZE*SIZE*SIZE+1)*threadIdx.x) = anscount;
	return;
}

void set_end_value(tree* root, int* board){
	int white = 9;
	int black = 0;
	for(int x = 0; x < SIZE; x++){
		for(int y = 0; y < SIZE; y++){
			switch(board[x] >> (2*y) & 0x00000003){
				case 2:
					black++;
					root = root->next_block[2];
					break;
				case 3:
					white++;
					root = root->next_block[3];
					break;
				default:
					root = root->next_block[0];
					break;
			}
		}
	}
	if(black > white){
		root->leaf->black_value = 1;
		root->leaf->white_value = 0;
	}
	else if(white > black){
		root->leaf->black_value = 0;
		root->leaf->white_value = 1;
	}
	else{
		root->leaf->black_value = 1;
		root->leaf->white_value = 1;
	}
	return;
}

leaves* get_leaf_pointer(tree* root, int* board){
	for(int x = 0; x < SIZE; x++){
		for(int y = 0; y < SIZE; y++){
			root = root->next_block[board[x] >> (2*y) & 0x00000003];
		}
	}
	return root->leaf;
}

void search_recu(int thread, int* cuda_data_in, int color, tree* root, int level){
	//cout << "in." << endl;
	int* cuda_data_out;
	hipMallocManaged(&cuda_data_out, sizeof(int)*(SIZE*SIZE*SIZE+1)*thread);
	cuda_check <<<1, thread>>> (cuda_data_in, cuda_data_out, SIZE, color);
	int* next_level_data;
	hipMallocManaged(&next_level_data, sizeof(int)*SIZE*maxthreads);
	hipDeviceSynchronize();
	int data_count = 0;
	for(int thread_num = 0; thread_num < thread; thread_num++){
		for(int a = 0; a < *(cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num); a++){
			make_leaf(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a);
			if(get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->white_value == -1){
				memcpy(next_level_data+SIZE*data_count, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a, sizeof(int)*SIZE);
				//print(next_level_data+SIZE*data_count);
				data_count++;
				if(data_count == maxthreads){
					search_recu(maxthreads, next_level_data, 1-color, root, level+1);
					data_count = 0;
				}
			}
			else if(get_leaf_pointer(root, cuda_data_in+SIZE*thread_num) == get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)){
				memcpy(next_level_data+SIZE*data_count, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a, sizeof(int)*SIZE);
				//print(next_level_data+SIZE*data_count);
				data_count++;
				if(data_count == maxthreads){
					search_recu(maxthreads, next_level_data, 1-color, root, level+1);
					data_count = 0;
				}
			}
		}
	}
	if(data_count != 0){
		search_recu(data_count, next_level_data, 1-color, root, level+1);
	}
	for(int thread_num = 0; thread_num < thread; thread_num++){
		if(*(cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num) == 0){
			set_end_value(root, cuda_data_in+SIZE*thread_num);
			/*
			print(cuda_data_in+SIZE*thread_num);
			char meow;
			cin >> meow;
			*/
			endcount++;
		}
		else{
			get_leaf_pointer(root, cuda_data_in+SIZE*thread_num)->black_value = 1;
			for(int a = 0; a < *(cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num); a++){
				if(get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->white_value == 1){
					get_leaf_pointer(root, cuda_data_in+SIZE*thread_num)->black_value = 0;
					break;
				}
				if(get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->white_value == -1){
					
					cout << "ERROR! in white value" << endl;
					cout << get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->black_value << " " << get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->white_value << endl;
					print(cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a);
					char meow;
					cin >> meow;
					
					error_count++;
				}
			}
			get_leaf_pointer(root, cuda_data_in+SIZE*thread_num)->white_value = 1;
			for(int a = 0; a < *(cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num); a++){
				if(get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->black_value == 1){
					get_leaf_pointer(root, cuda_data_in+SIZE*thread_num)->white_value = 0;
					break;
				}
				if(get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->black_value == -1){
					
					cout << "ERROR! in black value" << endl;
					cout << get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->black_value << " " << get_leaf_pointer(root, cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a)->white_value << endl;
					print(cuda_data_out+(SIZE*SIZE*SIZE+1)*thread_num+1+SIZE*a);
					char meow;
					cin >> meow;
					
					error_count++;
				}
			}
		}
	}
	hipFree(&cuda_data_out);
	hipFree(&next_level_data);
	if(level >= SIZE*SIZE){
		//cout << level << " " << count_child(root, 0) << endl;
	}
	else{
		//cout << level << endl;
	}
	//cout << "return." << endl;
	return;
}
